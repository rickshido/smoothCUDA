#include "hip/hip_runtime.h"
/* 
Command syntax:
$convolution <input_image> <output_image>
*/


#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cv.h>
#include <highgui.h>
#include "convolution_v3.h"


/*----------------------------- CUDA KERNELS -------------------------------*/
__global__
void conv(int *inImage, int *outImage, int rows, int columns) 
{
	int soma        = 0;
	int maskSize    = 15;
	int padding     = (int) maskSize / 2;
	int initPadding = padding * (-1);
	int endPadding  = padding;
	int divisor     = maskSize * maskSize;
	int i           = threadIdx.y + blockDim.y * blockIdx.y;
	int j           = threadIdx.x + blockDim.x * blockIdx.x;
	int a,b;

	if( (i<(rows-1)) && (j<(columns-1)) && (i>0) && (j>0)) 
	{
		soma = 0;
		for(a=initPadding; a<=endPadding; a++)
		{
			for(b=initPadding; b<=endPadding; b++)
			{
				soma = soma + inImage[(j+a) + (i+b) * columns];
			}
		}
		
		outImage[j + i * columns] = (int) (soma/divisor);
	}
}
/*----------------------- END - CUDA KERNELS -------------------------------*/


int main(int argc, char *argv[]) 
{
	char *inputfile  = argv[1];
	char *outputfile = argv[2];

	IplImage* input  = cvLoadImage(inputfile, CV_LOAD_IMAGE_COLOR);
	IplImage* output;

	clock_t time;

	double time_taken;

	time = clock();

	output = convolution(input);

	time = clock() - time;

	time_taken = ((double)time)/CLOCKS_PER_SEC;
        printf("convolution took %f seconds to execute \n", time_taken);

	cvSaveImage(outputfile, output);

	if(!input)  cvReleaseImage(&input);
	if(!output) cvReleaseImage(&output);

	return 0;
}


IplImage *convolution(IplImage *image) 
{ 
	int *h_inMatrix, *b_outMatrix, *g_outMatrix, *r_outMatrix;
	int *d_inMatrix, *d_outMatrix;

	int buffer_size;
	int m, n;

	hipError_t err;

	m = image->height/2;
	n = image->width;

	buffer_size = sizeof(int) * image->width * image->height/2;

	dim3 DimGrid((n-1)/16+1,(m-1)/16+1,1);
	dim3 DimBlock(16,16,1);

	int i;


		//BLUE
		h_inMatrix = getMatrix(image, 0);
		b_outMatrix = emptyMatrix(image);

		hipMalloc((void**) &d_inMatrix, buffer_size);
		hipMalloc((void**) &d_outMatrix, buffer_size);

		hipMemcpy(d_inMatrix,  h_inMatrix,  buffer_size, hipMemcpyHostToDevice);
		hipMemcpy(d_outMatrix, b_outMatrix, buffer_size, hipMemcpyHostToDevice);

		conv<<<DimGrid, DimBlock>>>(d_inMatrix, d_outMatrix, image->height, image->width);	
	
		hipMemcpy(b_outMatrix, d_outMatrix, buffer_size, hipMemcpyDeviceToHost);

		hipFree(d_inMatrix);
		hipFree(d_outMatrix);
		free(h_inMatrix);

		//Green
		h_inMatrix = getMatrix(image, 1);
       	 	g_outMatrix = emptyMatrix(image);

        	hipMalloc((void**) &d_inMatrix, buffer_size);
        	hipMalloc((void**) &d_outMatrix, buffer_size);

        	hipMemcpy(d_inMatrix,  h_inMatrix,  buffer_size, hipMemcpyHostToDevice);
        	hipMemcpy(d_outMatrix, g_outMatrix, buffer_size, hipMemcpyHostToDevice);

        	conv<<<DimGrid, DimBlock>>>(d_inMatrix, d_outMatrix, image->height, image->width);

        	hipMemcpy(g_outMatrix, d_outMatrix, buffer_size, hipMemcpyDeviceToHost);

        	hipFree(d_inMatrix);
        	hipFree(d_outMatrix);
        	free(h_inMatrix);

		//RED
		h_inMatrix = getMatrix(image, 2);
        	r_outMatrix = emptyMatrix(image);

        	hipMalloc((void**) &d_inMatrix, buffer_size);
        	hipMalloc((void**) &d_outMatrix, buffer_size);

        	hipMemcpy(d_inMatrix,  h_inMatrix,  buffer_size, hipMemcpyHostToDevice);
        	hipMemcpy(d_outMatrix, r_outMatrix, buffer_size, hipMemcpyHostToDevice);

        	conv<<<DimGrid, DimBlock>>>(d_inMatrix, d_outMatrix, image->height, image->width);

        	hipMemcpy(r_outMatrix, d_outMatrix, buffer_size, hipMemcpyDeviceToHost);

        	hipFree(d_inMatrix);
        	hipFree(d_outMatrix);
        	free(h_inMatrix);


	return matrixToIpl(b_outMatrix, g_outMatrix, r_outMatrix, image->width, image->height);
}


void checkError(hipError_t err) 
{ 
	if(err != hipSuccess) 
	{
		printf("CUDA error:\n");
		printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
}


void printMatrix(int *mat, int rows, int columns) 
{
	int i,j;

	for(i=0; i<rows; i++) 
	{
		for(j=0; j<columns; j++) 
		{
			printf("%.2f ", mat[j + i*columns]);
		}
		printf("\n");
	}
}


IplImage *loadImage(char *path) 
{
	IplImage *image = cvLoadImage(path, -1);

	if(!image) 
	{
		printf("\nError on load image: %s", path);
		exit(EXIT_FAILURE);
	}

	return image;
}


void showImageProperties(IplImage *image) 
{
	if(image) 
	{
		printf("Width: %d\n", image->width);
		printf("Height: %d\n", image->height);
		printf("Channels: %d\n", image->nChannels);
	} else {
		printf("Image is NULL\n");
	}
}


int *getMatrix(IplImage *image, int channel) 
{
	int i, j;

	int *matrix = (int*) malloc(sizeof(int) * image->width * image->height);

	for( i = 0; i < image->height; i++ ) 
		for( j = 0; j < image->width; j++ ) 
			matrix[i * image->width + j] = cvGet2D(image, i, j).val[channel];

	return matrix;
}


int *emptyMatrix(IplImage *image)
{
	return (int*) malloc(sizeof(int) * image->width * image->height);
}


IplImage *matrixToIpl(int *b, int *g, int *r, int width, int height) 
{
	int i, j;
	CvScalar pixel;

	IplImage *image = cvCreateImage(cvSize(width, height), IPL_DEPTH_8U, 3);

	for( i = 0; i < height; i++ )
	{
		for( j = 0; j < width; j++ )
		{
			cvSet2D(image, i, j, cvScalar(b[i * width + j], g[i * width + j], r[i * width + j], 0));
		}
	}

	return image;
}
